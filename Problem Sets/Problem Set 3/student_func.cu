#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

__global__ void reduce_max_min(const float* const d_in, float* d_out, bool is_max=true)
{
	extern __shared__ float partial[];

	int tid = threadIdx.x;
	int idx = blockIdx.x *  blockDim.x + tid;

	partial[tid] = d_in[idx];
	// make sure all data in this block has loaded into shared memory
	__syncthreads();
	
	for(unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1){
		if(tid < stride){
			if(is_max)
				partial[tid] = max(partial[tid], partial[tid+stride]);	
			else
				partial[tid] = min(partial[tid], partial[tid+stride]);	
		}
		// make sure all operations at one stage are done!
		__syncthreads();
	}
	

	if(tid == 0)
		d_out[blockIdx.x] = partial[tid];
}

void reduce(const float* const d_in,float &min_logLum,float &max_logLum,const size_t numRows,const size_t numCols)
{

	const int BLOCK_SIZE = numCols;
	const int GRID_SIZE  = numRows;
		// declare GPU memory pointers
	float * d_intermediate, *d_max, *d_min;
		
	// allocate GPU memory
	hipMalloc((void **) &d_intermediate, GRID_SIZE*sizeof(float));
	hipMalloc((void **) &d_max, sizeof(float));
	hipMalloc((void **) &d_min, sizeof(float));

	// find maximum;
	// firstly, find the maximum in each block
	reduce_max_min<<<GRID_SIZE,BLOCK_SIZE, BLOCK_SIZE*sizeof(float)>>>(d_in, d_intermediate, true);
	// then, find the global maximum
	reduce_max_min<<<1, GRID_SIZE, GRID_SIZE*sizeof(float)>>>(d_intermediate, d_max, true);

	checkCudaErrors(hipMemset(d_intermediate,0,GRID_SIZE*sizeof(float)));
	// find minimum;
	// firstly, find the minimum in each block
	reduce_max_min<<<GRID_SIZE,BLOCK_SIZE, BLOCK_SIZE*sizeof(float)>>>(d_in, d_intermediate,false);
	// then, find the global minimum
	reduce_max_min<<<1, GRID_SIZE, GRID_SIZE*sizeof(float)>>>(d_intermediate, d_min, false);
	

	// transfer the output to CPU
	checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));

	// free GPU memory location
	checkCudaErrors(hipFree(d_intermediate));
	checkCudaErrors(hipFree(d_max));
	checkCudaErrors(hipFree(d_min));

	return;	
}


__global__ void hist(const float* const d_in, unsigned int * const d_out, const float logLumRange, const int min_logLum, const int numBins)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float num = d_in[idx];
	int bin_idx = (num - min_logLum)/logLumRange*numBins;
	if(bin_idx >= numBins)
		bin_idx--;
	atomicAdd(&(d_out[bin_idx]),1);
	
}


// Hillis Steele Scan
__global__ void prefixSum_HS(const unsigned int * const d_in, unsigned int * const d_out)
{

	extern __shared__ float partial[];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + tid;

	// make sure all data in this block are loaded into shared shared memory
	partial[tid] = d_in[idx];
	__syncthreads();
	
	for(unsigned int stride = 1; stride < blockDim.x; stride <<= 1){
		if(tid + stride < blockDim.x)
			partial[tid+stride] += partial[tid];
		// make sure all operations at one stage are done!
		__syncthreads();
	}

	// exclusive scan
	if(tid == 0)
		d_out[tid] = 0;
	else
		d_out[tid] = partial[tid-1];	
}


void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */


	
	// Step 1 : find minimum and maximum value
	reduce(d_logLuminance, min_logLum, max_logLum, numRows, numCols);

	// Step 2: find the range 
	float logLumRange = max_logLum - min_logLum;

	// Step 3 : generate a histogram of all the values
	// declare GPU memory pointers
	unsigned int  *d_bins;
	// allocate GPU memory
	checkCudaErrors(hipMalloc((void **) &d_bins, numBins*sizeof(unsigned int)));
	checkCudaErrors(hipMemset(d_bins,0,numBins*sizeof(unsigned int)));
	
	hist<<<numRows, numCols>>>(d_logLuminance, d_bins, logLumRange, min_logLum, numBins);
	
	// Step 4 : prefix sum
	prefixSum_HS<<<1, numBins, numBins*sizeof(unsigned int)>>>(d_bins, d_cdf);

	// free GPU memory allocation
	checkCudaErrors(hipFree(d_bins));
}
